#include "hip/hip_runtime.h"
#include "mpc.cuh"

static const int GROUP_PER_WARP = WARP_SIZE / PATH_NUM;
static const int GROUP_PER_BLOCK = GROUP_PER_WARP * COMMIT_WARP_PER_BLOCK;

// static const int COMMIT_THREAD_PER_BLOCK = WARP_SIZE * COMMIT_WARP_PER_BLOCK;
// static const int GROUP_PER_BLOCK = COMMIT_THREAD_PER_BLOCK / PATH_NUM;

// static const int COMMIT_BLOCK_PER_GRID = (NUM_ROUNDS - 1) / GROUP_PER_BLOCK + 1;

__constant__ uint32_t hA[5] = { 0x67452301, 0xEFCDAB89, 0x98BADCFE, 0x10325476,0xC3D2E1F0 };

__device__ void dMpcXor(int gpid, uint32_t *x, uint32_t *y, uint32_t *z) {
	z[gpid] = x[gpid] ^ y[gpid];
}

__device__ void dMpcXorOneCore(uint32_t *x, uint32_t *y, uint32_t *z)
{
	z[0] = x[0] ^ y[0];
	z[1] = x[1] ^ y[1];
	z[2] = x[2] ^ y[2];
}

__device__ void dMpcAnd(int gpid, uint32_t *x, uint32_t *y, uint32_t *z, unsigned char (*randomness)[RANDTAPE_SIZE], int* randCount, View* view, int* countY) {
	int gpid1 = (gpid + 1) % PATH_NUM;

	uint32_t r[2] = { getRandom32(randomness[gpid], *randCount), getRandom32(randomness[gpid1], *randCount) };
	*randCount += 4;

	z[gpid] = (x[gpid] & y[gpid1]) ^ (x[gpid1] & y[gpid]) ^ (x[gpid] & y[gpid]) ^ r[0] ^ r[1];

	view->y[*countY] = z[gpid];
	(*countY)++;
}

__device__ void dMpcAndOneCore(uint32_t *x, uint32_t *y, uint32_t *z, unsigned char (*randomness)[RANDTAPE_SIZE], int* randCount, View* views, int* countY)
{
	uint32_t r[3] = { getRandom32(randomness[0], *randCount), getRandom32(randomness[1], *randCount), getRandom32(randomness[2], *randCount)};
	*randCount += 4;
	uint32_t t[3] = { 0 };

	t[0] = (x[0] & y[1]) ^ (x[1] & y[0]) ^ (x[0] & y[0]) ^ r[0] ^ r[1];
	t[1] = (x[1] & y[2]) ^ (x[2] & y[1]) ^ (x[1] & y[1]) ^ r[1] ^ r[2];
	t[2] = (x[2] & y[0]) ^ (x[0] & y[2]) ^ (x[2] & y[2]) ^ r[2] ^ r[0];
	z[0] = t[0];
	z[1] = t[1];
	z[2] = t[2];

	views[0].y[*countY] = z[0];
	views[1].y[*countY] = z[1];
	views[2].y[*countY] = z[2];

	(*countY)++;
}

__device__ void dMpcNegate(int gpid, uint32_t *x, uint32_t *z) {
	z[gpid] = ~x[gpid];
}

__device__ void dMpcNegateOneCore(uint32_t *x, uint32_t *z)
{
	z[0] = ~x[0];
	z[1] = ~x[1];
	z[2] = ~x[2];
}

__device__ void dMpcAdd(int bgid, int gpid, uint32_t *x, uint32_t *y, uint32_t *z, unsigned char (*randomness)[RANDTAPE_SIZE], int* randCount, View* view, int* countY) {
	int gpid1 = (gpid + 1) % PATH_NUM;

	uint32_t c = 0;
	uint32_t r[2] = { getRandom32(randomness[gpid], *randCount), getRandom32(randomness[gpid1], *randCount) };
	*randCount += 4;

	__shared__ uint8_t as[GROUP_PER_BLOCK][PATH_NUM];
	__shared__ uint8_t bs[GROUP_PER_BLOCK][PATH_NUM];

	uint8_t* a = as[bgid];
	uint8_t* b = bs[bgid];

	uint8_t t = 0;

	for (int i = 0; i < 31; i++) {
		a[gpid] = GETBIT(x[gpid] ^ c, i);
		b[gpid] = GETBIT(y[gpid] ^ c, i);

		t = (a[gpid] & b[gpid1]) ^ (a[gpid1] & b[gpid]) ^ (a[gpid] & b[gpid]) ^ GETBIT(r[1], i) ^ GETBIT(r[0], i) ^ t;
		
		c |= (t << (i + 1));
	}

	z[gpid] = x[gpid] ^ y[gpid] ^ c;

	view->y[*countY] = c;
	*countY += 1;
}


__device__ void dMpcAddOneCore(uint32_t *x, uint32_t *y, uint32_t *z, unsigned char (*randomness)[RANDTAPE_SIZE], int* randCount, View* views, int* countY)
{
	uint32_t c[3] = { 0 };
	uint32_t r[3] = { getRandom32(randomness[0], *randCount), getRandom32(randomness[1], *randCount), getRandom32(randomness[2], *randCount)};
	*randCount += 4;

	uint8_t a[3], b[3];
	uint8_t t;

	for(int i=0;i<31;i++)
	{
		a[0]=GETBIT(x[0]^c[0],i);
		a[1]=GETBIT(x[1]^c[1],i);
		a[2]=GETBIT(x[2]^c[2],i);

		b[0]=GETBIT(y[0]^c[0],i);
		b[1]=GETBIT(y[1]^c[1],i);
		b[2]=GETBIT(y[2]^c[2],i);

		t = (a[0]&b[1]) ^ (a[1]&b[0]) ^ GETBIT(r[1],i);
		SETBIT(c[0],i+1, t ^ (a[0]&b[0]) ^ GETBIT(c[0],i) ^ GETBIT(r[0],i));

		t = (a[1]&b[2]) ^ (a[2]&b[1]) ^ GETBIT(r[2],i);
		SETBIT(c[1],i+1, t ^ (a[1]&b[1]) ^ GETBIT(c[1],i) ^ GETBIT(r[1],i));

		t = (a[2]&b[0]) ^ (a[0]&b[2]) ^ GETBIT(r[0],i);
		SETBIT(c[2],i+1, t ^ (a[2]&b[2]) ^ GETBIT(c[2],i) ^ GETBIT(r[2],i));
	}

	z[0]=x[0]^y[0]^c[0];
	z[1]=x[1]^y[1]^c[1];
	z[2]=x[2]^y[2]^c[2];

	views[0].y[*countY] = c[0];
	views[1].y[*countY] = c[1];
	views[2].y[*countY] = c[2];
	*countY += 1;

}


__device__ void dMpcAddK(int bgid, int gpid, uint32_t *x, uint32_t y, uint32_t *z, unsigned char (*randomness)[RANDTAPE_SIZE], int* randCount, View* view, int* countY) {
	int gpid1 = (gpid + 1) % PATH_NUM;

	uint32_t c = 0;
	uint32_t r[2] = { getRandom32(randomness[gpid], *randCount), getRandom32(randomness[gpid1], *randCount) };
	*randCount += 4;

	__shared__ uint8_t as[GROUP_PER_BLOCK][PATH_NUM];
	__shared__ uint8_t bs[GROUP_PER_BLOCK][PATH_NUM];

	uint8_t* a = as[bgid];
	uint8_t* b = bs[bgid];

	uint8_t t = 0;
	
	for (int i = 0; i < 31; i++) {
		a[gpid] = GETBIT(x[gpid] ^ c, i);
		b[gpid] = GETBIT(y ^ c, i);

		t = (a[gpid] & b[gpid1]) ^ (a[gpid1] & b[gpid]) ^ (a[gpid] & b[gpid]) ^ GETBIT(r[1], i) ^ GETBIT(r[0], i) ^ t;

		c |= (t << (i + 1));
	}


	z[gpid] = x[gpid] ^ y ^ c;

	view->y[*countY] = c;
	*countY += 1;
}


__device__ void dMpcAddKOneCore(uint32_t *x, uint32_t y, uint32_t *z, unsigned char (*randomness)[RANDTAPE_SIZE], int* randCount, View* views, int* countY)
{
	uint32_t c[3] = { 0 };
	uint32_t r[3] = { getRandom32(randomness[0], *randCount), getRandom32(randomness[1], *randCount), getRandom32(randomness[2], *randCount)};
	*randCount += 4;

	uint8_t a[3], b[3];
	uint8_t t;

	for(int i=0;i<31;i++)
	{
		a[0]=GETBIT(x[0]^c[0],i);
		a[1]=GETBIT(x[1]^c[1],i);
		a[2]=GETBIT(x[2]^c[2],i);

		b[0]=GETBIT(y^c[0],i);
		b[1]=GETBIT(y^c[1],i);
		b[2]=GETBIT(y^c[2],i);

		t = (a[0]&b[1]) ^ (a[1]&b[0]) ^ GETBIT(r[1],i);
		SETBIT(c[0],i+1, t ^ (a[0]&b[0]) ^ GETBIT(c[0],i) ^ GETBIT(r[0],i));

		t = (a[1]&b[2]) ^ (a[2]&b[1]) ^ GETBIT(r[2],i);
		SETBIT(c[1],i+1, t ^ (a[1]&b[1]) ^ GETBIT(c[1],i) ^ GETBIT(r[1],i));

		t = (a[2]&b[0]) ^ (a[0]&b[2]) ^ GETBIT(r[0],i);
		SETBIT(c[2],i+1, t ^ (a[2]&b[2]) ^ GETBIT(c[2],i) ^ GETBIT(r[2],i));
	}

	z[0]=x[0]^y^c[0];
	z[1]=x[1]^y^c[1];
	z[2]=x[2]^y^c[2];


	views[0].y[*countY] = c[0];
	views[1].y[*countY] = c[1];
	views[2].y[*countY] = c[2];
	*countY += 1;
}


__device__ void dMpcRightRotate(int gpid, uint32_t *x, int i, uint32_t *z) {
	z[gpid] = RIGHTROTATE(x[gpid], i);
}

__device__ void dMpcRightRotateOneCore(uint32_t *x, int i, uint32_t *z)
{
	z[0] = RIGHTROTATE(x[0], i);
	z[1] = RIGHTROTATE(x[1], i);
	z[2] = RIGHTROTATE(x[2], i);
}

__device__ void dMpcLeftRotate(int gpid, uint32_t *x, int i, uint32_t *z) {
	z[gpid] = LEFTROTATE(x[gpid], i);
}



__device__ void dMpcLeftRotateOneCore(uint32_t *x, int i, uint32_t *z)
{
	z[0] = LEFTROTATE(x[0], i);
	z[1] = LEFTROTATE(x[1], i);
	z[2] = LEFTROTATE(x[2], i);
}


__device__ void dMpcRightShift(int gpid, uint32_t *x, int i, uint32_t *z) {
	z[gpid] = x[gpid] >> i;
}

__device__ void dMpcRightShiftOneCore(uint32_t *x, int i, uint32_t *z)
{
	z[0] = x[0] >> i;
	z[1] = x[1] >> i;
	z[2] = x[2] >> i;
}

__device__ void dMpcMAJ(int bgid, int gpid, uint32_t *a, uint32_t *b, uint32_t *c, uint32_t *z, unsigned char (*randomness)[RANDTAPE_SIZE], int* randCount, View* view, int* countY) {
	__shared__ uint32_t t0s[GROUP_PER_BLOCK][PATH_NUM];
	__shared__ uint32_t t1s[GROUP_PER_BLOCK][PATH_NUM];

	uint32_t* t0 = t0s[bgid];
	uint32_t* t1 = t1s[bgid];

	dMpcXor(gpid, a, b, t0);
	dMpcXor(gpid, a, c, t1);
	dMpcAnd(gpid, t0, t1, z, randomness, randCount, view, countY);
	dMpcXor(gpid, z, a, z);
}

__device__ void dMpcMAJOneCore(uint32_t *a, uint32_t *b, uint32_t *c, uint32_t *z, unsigned char (*randomness)[RANDTAPE_SIZE], int* randCount, View* views, int* countY)
{
	uint32_t t0[3];
	uint32_t t1[3];

	dMpcXorOneCore(a, b, t0);
	dMpcXorOneCore(a, c, t1);
	dMpcAndOneCore(t0, t1, z, randomness, randCount, views, countY);
	dMpcXorOneCore(z, a, z);
}

__device__ void dMpcSha1OneCore(unsigned char (*results)[32], unsigned char* input[3], int numBits, unsigned char (*randomness)[RANDTAPE_SIZE], View views[3], int* countY)
{
	int randCount = 0;
	int chars = numBits >> 3;
	unsigned char chunks[3][64];
	uint32_t w[80][3];

	for(int i=0; i<3 ; i++)
	{
		dMemcpy(chunks[i], input[i], chars);
		chunks[i][chars] = 0x80;

		dMemset(chunks[i] + chars + 1, 61 - chars, 0);
		chunks[i][62] = numBits >> 8;
		chunks[i][63] = numBits;

		dMemcpy(views[i].x, chunks[i], 64);

		for (int j = 0; j < 16; j++) 
		{
			w[j][i] = (chunks[i][j * 4] << 24) | (chunks[i][j * 4 + 1] << 16)
							| (chunks[i][j * 4 + 2] << 8) | chunks[i][j * 4 + 3];
		}
	}

	uint32_t temp[3];
	uint32_t t0[3];

	for (int j = 16; j < 80; j++) {
		dMpcXorOneCore(w[j-3], w[j-8], temp);
		dMpcXorOneCore(temp, w[j-14], temp);
		dMpcXorOneCore(temp, w[j-16], temp);
		dMpcLeftRotateOneCore(temp,1,w[j]);
	}

	uint32_t a[3] = { hA[0],hA[0],hA[0] };
	uint32_t b[3] = { hA[1],hA[1],hA[1] };
	uint32_t c[3] = { hA[2],hA[2],hA[2] };
	uint32_t d[3] = { hA[3],hA[3],hA[3] };
	uint32_t e[3] = { hA[4],hA[4],hA[4] };
	uint32_t f[3];
	uint32_t k;

	for (int i = 0; i < 80; i++) {
		if(i <= 19) {
			//f = d ^ (b & (c ^ d))
			dMpcXorOneCore(c,d,f);
			dMpcAndOneCore(b, f, f, randomness, &randCount, views, countY);
			dMpcXorOneCore(d,f,f);
			k = 0x5A827999;
		}
		else if(i <= 39) {
			dMpcXorOneCore(b,c,f);
			dMpcXorOneCore(d,f,f);
			k = 0x6ED9EBA1;
		}
		else if(i <= 59) {
			//f = MAJ(b,c,d)

			dMpcMAJOneCore(b,c,d,f,randomness, &randCount, views, countY);

			k = 0x8F1BBCDC;
		}
		else {
			dMpcXorOneCore(b,c,f);
			dMpcXorOneCore(d,f,f);
			k = 0xCA62C1D6;
		}

		//temp = (a leftrotate 5) + f + e + k + w[i]
		dMpcLeftRotateOneCore(a,5,temp);
		dMpcAddOneCore(f,temp,temp,randomness, &randCount, views, countY);
		dMpcAddOneCore(e,temp,temp,randomness, &randCount, views, countY);
		dMpcAddKOneCore(temp,k,temp,randomness, &randCount, views, countY);
		dMpcAddOneCore(w[i],temp,temp,randomness, &randCount, views, countY);

		dMemcpy(e, d, sizeof(uint32_t) * 3);
		dMemcpy(d, c, sizeof(uint32_t) * 3);
		dMpcLeftRotateOneCore(b,30,c);
		dMemcpy(b, a, sizeof(uint32_t) * 3);
		dMemcpy(a, temp, sizeof(uint32_t) * 3);
	}

	uint32_t hHa[5][3] = { { hA[0],hA[0],hA[0]  }, { hA[1],hA[1],hA[1] }, { hA[2],hA[2],hA[2] }, { hA[3],hA[3],hA[3] },
			{ hA[4],hA[4],hA[4] }};

	dMpcAddOneCore(hHa[0], a, hHa[0], randomness, &randCount, views, countY);
	dMpcAddOneCore(hHa[1], b, hHa[1], randomness, &randCount, views, countY);
	dMpcAddOneCore(hHa[2], c, hHa[2], randomness, &randCount, views, countY);
	dMpcAddOneCore(hHa[3], d, hHa[3], randomness, &randCount, views, countY);
	dMpcAddOneCore(hHa[4], e, hHa[4], randomness, &randCount, views, countY);

	for (int i = 0; i < 5; i++) {
		dMpcRightShiftOneCore(hHa[i], 24, t0);
		results[0][i * 4] = t0[0];
		results[1][i * 4] = t0[1];
		results[2][i * 4] = t0[2];
		dMpcRightShiftOneCore(hHa[i], 16, t0);
		results[0][i * 4 + 1] = t0[0];
		results[1][i * 4 + 1] = t0[1];
		results[2][i * 4 + 1] = t0[2];
		dMpcRightShiftOneCore(hHa[i], 8, t0);
		results[0][i * 4 + 2] = t0[0];
		results[1][i * 4 + 2] = t0[1];
		results[2][i * 4 + 2] = t0[2];

		results[0][i * 4 + 3] = hHa[i][0];
		results[1][i * 4 + 3] = hHa[i][1];
		results[2][i * 4 + 3] = hHa[i][2];
	}
}

__device__ void dMpcSha1(int bgid, int gpid, unsigned char* result, unsigned char* input, int numBits, unsigned char (*randomness)[RANDTAPE_SIZE], View* view, int* countY) {
	int randCount = 0;
	int chars = numBits >> 3;

	unsigned char chunk[64];

	__shared__ uint32_t ws[GROUP_PER_BLOCK][80][PATH_NUM];

	uint32_t(*w)[PATH_NUM] = ws[bgid];

	dMemcpy(chunk, input, chars);
	chunk[chars] = 0x80;

	dMemset(chunk + chars + 1, 61 - chars, 0);

	chunk[62] = numBits >> 8;
	chunk[63] = numBits;

	dMemcpy(view->x, chunk, 64);

	for (int j = 0; j < 16; j++) 
		w[j][gpid] = (chunk[j * 4] << 24)     |    
					 (chunk[j * 4 + 1] << 16) | 
					 (chunk[j * 4 + 2] << 8)  | 
					 (chunk[j * 4 + 3]);
	

	__shared__ uint32_t temps[GROUP_PER_BLOCK][PATH_NUM];
	__shared__ uint32_t t0s[GROUP_PER_BLOCK][PATH_NUM];

	uint32_t* temp = temps[bgid];
	uint32_t* t0 = t0s[bgid];

	for (int j = 16; j < 80; j++) {
		dMpcXor(gpid, w[j - 3], w[j - 8], temp);
		dMpcXor(gpid, temp, w[j - 14], temp);
		dMpcXor(gpid, temp, w[j - 16], temp);
		dMpcLeftRotate(gpid, temp, 1, w[j]);
	}

	__shared__ uint32_t as[GROUP_PER_BLOCK][PATH_NUM];
	__shared__ uint32_t bs[GROUP_PER_BLOCK][PATH_NUM];
	__shared__ uint32_t cs[GROUP_PER_BLOCK][PATH_NUM];
	__shared__ uint32_t ds[GROUP_PER_BLOCK][PATH_NUM];
	__shared__ uint32_t es[GROUP_PER_BLOCK][PATH_NUM];
	__shared__ uint32_t fs[GROUP_PER_BLOCK][PATH_NUM];

	uint32_t *a = as[bgid];
	uint32_t *b = bs[bgid];
	uint32_t *c = cs[bgid];
	uint32_t *d = ds[bgid];
	uint32_t *e = es[bgid];
	uint32_t *f = fs[bgid];

	uint32_t k;

	a[gpid] = hA[0];
	b[gpid] = hA[1];
	c[gpid] = hA[2];
	d[gpid] = hA[3];
	e[gpid] = hA[4];

	for (int i = 0; i < 80; i++) {
		if (i <= 19) {
			//f = d ^ (b & (c ^ d))
			dMpcXor(gpid, c, d, f);
			dMpcAnd(gpid, b, f, f, randomness, &randCount, view, countY);
			dMpcXor(gpid, d, f, f);
			k = 0x5A827999;
		}
		else if (i <= 39) {
			dMpcXor(gpid, b, c, f);
			dMpcXor(gpid, d, f, f);
			k = 0x6ED9EBA1;
		}
		else if (i <= 59) {
			//f = MAJ(b,c,d)
			dMpcMAJ(bgid, gpid, b, c, d, f, randomness, &randCount, view, countY);
			k = 0x8F1BBCDC;
		}
		else {
			dMpcXor(gpid, b, c, f);
			dMpcXor(gpid, d, f, f);
			k = 0xCA62C1D6;
		}

		//temp = (a leftrotate 5) + f + e + k + w[i]
		dMpcLeftRotate(gpid, a, 5, temp);
		dMpcAdd(bgid, gpid, f, temp, temp, randomness, &randCount, view, countY);
		dMpcAdd(bgid, gpid, e, temp, temp, randomness, &randCount, view, countY);
		dMpcAddK(bgid, gpid, temp, k, temp, randomness, &randCount, view, countY);
		dMpcAdd(bgid, gpid, w[i], temp, temp, randomness, &randCount, view, countY);

		e[gpid] = d[gpid];
		d[gpid] = c[gpid];
		dMpcLeftRotate(gpid, b, 30, c);
		b[gpid] = a[gpid];
		a[gpid] = temp[gpid];
	}

	__shared__ uint32_t hHas[GROUP_PER_BLOCK][5][PATH_NUM];

	uint32_t (*hHa)[PATH_NUM] = hHas[bgid];

	for (int i = 0; i < 5; i++)
		hHa[i][gpid] = hA[i];

	dMpcAdd(bgid, gpid, hHa[0], a, hHa[0], randomness, &randCount, view, countY);
	dMpcAdd(bgid, gpid, hHa[1], b, hHa[1], randomness, &randCount, view, countY);
	dMpcAdd(bgid, gpid, hHa[2], c, hHa[2], randomness, &randCount, view, countY);
	dMpcAdd(bgid, gpid, hHa[3], d, hHa[3], randomness, &randCount, view, countY);
	dMpcAdd(bgid, gpid, hHa[4], e, hHa[4], randomness, &randCount, view, countY);

	for (int i = 0; i < 5; i++) {
		dMpcRightShift(gpid, hHa[i], 24, t0);
		result[i * 4] = t0[gpid];

		dMpcRightShift(gpid, hHa[i], 16, t0);
		result[i * 4 + 1] = t0[gpid];

		dMpcRightShift(gpid, hHa[i], 8, t0);
		result[i * 4 + 2] = t0[gpid];

		result[i * 4 + 3] = hHa[i][gpid];
	}
}

__device__ void output(View* v, void* result) {
	dMemcpy(result, &v->y[YSIZE - 5], 20);
}








